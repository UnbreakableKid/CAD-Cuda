#include "hip/hip_runtime.h"
/**
 * Hervé Paulino
 * tD7Ymjz5m$$RrxFN
 */

#include <nbody/cuda_nbody_all_pairs.h>

static constexpr int thread_block_size = 512;

static constexpr int nStreams = 3;

namespace cadlabs {

    cuda_nbody_all_pairs::cuda_nbody_all_pairs(
            const int number_particles,
            const float t_final,
            const unsigned number_of_threads,
            const universe_t universe,
            const unsigned universe_seed) :
            nbody(number_particles, t_final, universe, universe_seed),
            number_blocks((number_particles + thread_block_size - 1) / thread_block_size) {


        hipMalloc((void **) &gpu_particles, number_particles * sizeof(particle_t));
    }

    cuda_nbody_all_pairs::~cuda_nbody_all_pairs() {
        hipFree(gpu_particles);
    }

#pragma region
#if __CUDA_ARCH__ < 600

    __device__ double atomicAdd(double *address, double val) {
        unsigned long long int *address_as_ull =
                (unsigned long long int *) address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(val +
                                                 __longlong_as_double(assumed)));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }

#endif


    __device__ static float atomicMax(float *address, float val) {
        int *address_as_i = (int *) address;
        int old = *address_as_i, assumed;
        do {
            assumed = old;
            old = ::atomicCAS(address_as_i, assumed,
                              __float_as_int(::fmaxf(val, __int_as_float(assumed))));
        } while (assumed != old);
        return __int_as_float(old);
    }

    __device__ static double atomicMax(double *address, double val) {
        unsigned long long int *address_as_ull =
                (unsigned long long int *) address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(::fmax(val ,
                                                 __longlong_as_double(assumed))));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }
#pragma endregion

    //! Function With paralelized adding, using global memory and atomic adds, takes too long
    //! \param particles
    //! \param number_particles
    //! \param pi
    __global__ void addingWithGlobalMemory(particle_t *particles, const unsigned number_particles, particle_t *pi) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < number_particles) {

            particle_t *pj = &particles[index];

            double x_sep, y_sep, dist_sq, grav_base;
            x_sep = pj->x_pos - pi->x_pos;
            y_sep = pj->y_pos - pi->y_pos;
            dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

            /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
            grav_base = GRAV_CONSTANT * (pi->mass) * (pj->mass) / dist_sq;

            double x = grav_base * x_sep;
            double y = grav_base * y_sep;

            atomicAdd(&(pi->x_force), x);
            atomicAdd(&(pi->y_force), y);
        }
    }

    __global__ void test(particle_t *particles, const unsigned number_particles, particle_t *pi) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int lindex = threadIdx.x;

        extern __shared__ particle_t temp[]; //make sure every thread has loaded

        temp[lindex] = particles[index];
        __syncthreads();

        if (index < number_particles) {

            particle_t *pj = &temp[lindex];

            double x_sep, y_sep, dist_sq, grav_base;
            x_sep = pj->x_pos - pi->x_pos;
            y_sep = pj->y_pos - pi->y_pos;
            dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

            /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
            grav_base = GRAV_CONSTANT * (pi->mass) * (pj->mass) / dist_sq;

            double x = grav_base * x_sep;
            double y = grav_base * y_sep;

            atomicAdd(&(pi->x_force), x);
            atomicAdd(&(pi->y_force), y);

        }
    }

    __global__ void nbody_kernel(particle_t *particles, const unsigned number_particles, int offset) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < number_particles) {

            particle_t *pi = &particles[index]; //fazer em mem partilhada

            pi->x_force = 0;
            pi->y_force = 0;

            for (int j = 0; j < number_particles; j++) {
                particle_t *p = &particles[j];
                /* compute the force of particle j on particle i */
                double x_sep, y_sep, dist_sq, grav_base;

                x_sep = p->x_pos - pi->x_pos;
                y_sep = p->y_pos - pi->y_pos;
                dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

                /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
                grav_base = GRAV_CONSTANT * (pi->mass) * (p->mass) / dist_sq;

                pi->x_force += grav_base * x_sep;
                pi->y_force += grav_base * y_sep;            }

        }

    }

    void cuda_nbody_all_pairs::calculate_forces() {
        /* First calculate force for particles. */

        particle_t *out;
        hipMalloc((void **) &out, number_particles * sizeof(particle_t));


        hipStream_t stream1, stream2, stream3;
        hipStreamCreate ( &stream1) ;
        hipStreamCreate ( &stream2) ;
        hipStreamCreate ( &stream3) ;

        hipStream_t stream[nStreams] = {stream1, stream2, stream3};

        int streamSize =number_particles;

//        for (int i = 0; i < nStreams; ++i) {
//            int offset = i * streamSize;
//            hipMemcpyAsync(&gpu_particles[offset], &particles[offset], number_particles * sizeof(particle_t), hipMemcpyHostToDevice, stream[i]);
//            nbody_kernel<<<streamSize/number_blocks, thread_block_size, 0, stream[i]>>>( gpu_particles, number_particles,offset);
//            hipMemcpyAsync(&particles[offset], &gpu_particles[offset], number_particles * sizeof(particle_t), hipMemcpyDeviceToHost, stream[i]);
//        }

        hipMemcpy(gpu_particles, particles, number_particles * sizeof(particle_t), hipMemcpyHostToDevice);
        nbody_kernel<<<number_blocks, thread_block_size>>>(gpu_particles, number_particles, 0);

//        for (int i = 0; i <number_particles; i++){
//
//            test<<<number_blocks, thread_block_size>>>(gpu_particles, number_particles, &gpu_particles[i]);
//        }
//
//        hipMemcpy(particles, gpu_particles, number_particles * sizeof(particle_t), hipMemcpyDeviceToHost);

    }


    //@ statistics variables

    __global__ void all_pairs_kernel(particle_t *particles, const int number_particles, double step, double *speeds, double *accs) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

//        printf("TEMP = %f\n", step);

        if (index < number_particles) {
            particle_t *pi = &particles[index];

            pi->x_pos += (pi->x_vel) * step;
            pi->y_pos += (pi->y_vel) * step;
            double x_acc = pi->x_force / pi->mass;
            double y_acc = pi->y_force / pi->mass;

            pi->x_vel += x_acc * step;
            pi->y_vel += y_acc * step;

            /* compute statistics */
            double cur_acc = (x_acc * x_acc + y_acc * y_acc);
            cur_acc = sqrt(cur_acc);
            double speed_sq = (pi->x_vel) * (pi->x_vel) + (pi->y_vel) * (pi->y_vel);
            double cur_speed = sqrt(speed_sq);

            accs[index] = cur_acc;
            speeds[index] = cur_speed;

        }
    }

    double * cuda_nbody_all_pairs::move_all_particles(double step) {
//
        double *speeds;
        double *d_speeds;
        speeds = (double *)malloc(sizeof(double )* number_particles);

        hipMalloc((void**)&d_speeds,(sizeof(double )* number_particles));

        double *accs;
        double *d_accs;
        accs = (double *)malloc(sizeof(double )* number_particles);

        hipMalloc((void**)&d_accs,(sizeof(double )* number_particles));

        all_pairs_kernel<<<number_blocks, thread_block_size>>>(gpu_particles,number_particles,step, d_speeds,d_accs);

        double speed = 0;
        double acc = 0;

        hipMemcpy(accs, d_accs, number_particles * sizeof(double ), hipMemcpyDeviceToHost);
        hipMemcpy(speeds, d_speeds, number_particles * sizeof(double ), hipMemcpyDeviceToHost);
        hipMemcpy(particles, gpu_particles, number_particles * sizeof(particle_t), hipMemcpyDeviceToHost);

        for (int i= 0; i < number_particles; i++){
            speed = MAX(speed, speeds[i]);
            acc = MAX(acc, accs[i]);
        }

        hipFree(d_speeds);
        hipFree(d_accs);
        return new double[2]{acc, speed};
    }

    void cuda_nbody_all_pairs::print_all_particles(std::ostream &out) {
        nbody::print_all_particles(out);
    }

}

