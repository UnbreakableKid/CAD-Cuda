#include "hip/hip_runtime.h"
/**
 * Hervé Paulino
 * tD7Ymjz5m$$RrxFN
 */

#include <nbody/cuda_nbody_all_pairs.h>

static constexpr int thread_block_size = 512;

static constexpr int nStreams = 4;

namespace cadlabs {

    cuda_nbody_all_pairs::cuda_nbody_all_pairs(
            const int number_particles,
            const float t_final,
            const unsigned number_of_threads,
            const universe_t universe,
            const unsigned universe_seed) :
            nbody(number_particles, t_final, universe, universe_seed),
            number_blocks((number_particles + thread_block_size - 1) / thread_block_size) {


        hipMalloc((void **) &gpu_particles, number_particles * sizeof(particle_t));
    }

    cuda_nbody_all_pairs::~cuda_nbody_all_pairs() {
        hipFree(gpu_particles);
    }

#pragma region
#if __CUDA_ARCH__ < 600

    __device__ double atomicAdd(double *address, double val) {
        unsigned long long int *address_as_ull =
                (unsigned long long int *) address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(val +
                                                 __longlong_as_double(assumed)));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }

#endif


    __device__ static float atomicMax(float *address, float val) {
        int *address_as_i = (int *) address;
        int old = *address_as_i, assumed;
        do {
            assumed = old;
            old = ::atomicCAS(address_as_i, assumed,
                              __float_as_int(::fmaxf(val, __int_as_float(assumed))));
        } while (assumed != old);
        return __int_as_float(old);
    }

    __device__ static double atomicMax(double *address, double val) {
        unsigned long long int *address_as_ull =
                (unsigned long long int *) address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(::fmax(val ,
                                                 __longlong_as_double(assumed))));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }
#pragma endregion

    //! Function With paralelized adding, using global memory and atomic adds, takes too long
    //! \param particles
    //! \param number_particles
    //! \param pi
    __global__ void addingWithGlobalMemory(particle_t *particles, const unsigned number_particles, particle_t *pi) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < number_particles) {

            particle_t *pj = &particles[index];

            double x_sep, y_sep, dist_sq, grav_base;
            x_sep = pj->x_pos - pi->x_pos;
            y_sep = pj->y_pos - pi->y_pos;
            dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

            /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
            grav_base = GRAV_CONSTANT * (pi->mass) * (pj->mass) / dist_sq;

            double x = grav_base * x_sep;
            double y = grav_base * y_sep;

            atomicAdd(&(pi->x_force), x);
            atomicAdd(&(pi->y_force), y);
        }
    }

    __global__ void test(particle_t *particles, const unsigned number_particles, particle_t *pi) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int lindex = threadIdx.x;

        extern __shared__ particle_t temp[]; //make sure every thread has loaded

        temp[lindex] = particles[index];
        __syncthreads();

        if (index < number_particles) {

            particle_t *pj = &temp[lindex];

            double x_sep, y_sep, dist_sq, grav_base;
            x_sep = pj->x_pos - pi->x_pos;
            y_sep = pj->y_pos - pi->y_pos;
            dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

            /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
            grav_base = GRAV_CONSTANT * (pi->mass) * (pj->mass) / dist_sq;

            double x = grav_base * x_sep;
            double y = grav_base * y_sep;

            atomicAdd(&(pi->x_force), x);
            atomicAdd(&(pi->y_force), y);

        }
    }

    __global__ void nbody_kernel(particle_t *particles, const unsigned number_particles, int offset) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < number_particles) {

            particle_t *pi = &particles[index]; //fazer em mem partilhada

            pi->x_force = 0;
            pi->y_force = 0;

            for (int j = 0; j < number_particles; j++) {
                particle_t *p = &particles[j];
                /* compute the force of particle j on particle i */
                double x_sep, y_sep, dist_sq, grav_base;

                x_sep = p->x_pos - pi->x_pos;
                y_sep = p->y_pos - pi->y_pos;
                dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

                /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
                grav_base = GRAV_CONSTANT * (pi->mass) * (p->mass) / dist_sq;

                pi->x_force += grav_base * x_sep;
                pi->y_force += grav_base * y_sep;
                }

        }

    }

    void cuda_nbody_all_pairs::calculate_forces() {
        /* First calculate force for particles. */

        hipMemcpy(gpu_particles, particles, number_particles * sizeof(particle_t), hipMemcpyHostToDevice);
        nbody_kernel<<<number_blocks, thread_block_size>>>(gpu_particles, number_particles, 0);

        for (int i = 0; i <number_particles; i++){

            addingWithGlobalMemory<<<number_blocks, thread_block_size>>>(gpu_particles, number_particles, &gpu_particles[i]);
        }
        hipMemcpy(particles, gpu_particles, number_particles * sizeof(particle_t), hipMemcpyDeviceToHost);

    }


    //@ statistics variables

    __global__ void all_pairs_kernel(particle_t *particles, const int number_particles, double step, double *speeds, double *accs) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;

//        printf("TEMP = %f\n", step);

        if (index < number_particles) {
            particle_t *pi = &particles[index];

            pi->x_pos += (pi->x_vel) * step;
            pi->y_pos += (pi->y_vel) * step;
            double x_acc = pi->x_force / pi->mass;
            double y_acc = pi->y_force / pi->mass;

            pi->x_vel += x_acc * step;
            pi->y_vel += y_acc * step;

            /* compute statistics */
            double cur_acc = (x_acc * x_acc + y_acc * y_acc);
            cur_acc = sqrt(cur_acc);
            double speed_sq = (pi->x_vel) * (pi->x_vel) + (pi->y_vel) * (pi->y_vel);
            double cur_speed = sqrt(speed_sq);

            accs[index] = cur_acc;
            speeds[index] = cur_speed;

        }
    }

    double * cuda_nbody_all_pairs::move_all_particles(double step) {

        return nbody::move_all_particles(step);
    }

    void cuda_nbody_all_pairs::print_all_particles(std::ostream &out) {
        nbody::print_all_particles(out);
    }

}

